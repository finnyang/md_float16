#include "hip/hip_runtime.h"

#include "hip/hip_fp16.h"
#include <stdio.h>
#include <iostream>
using namespace std;
 
#define CHECK(call) \
{ \
	const hipError_t error = call; \
	if (error != hipSuccess) \
	{ \
		printf("Error: %s: %d, ", __FILE__, __LINE__); \
		printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
		system("pause"); \
	} \
}
 
__global__ void myHalf2Add(half2 *a, half2 *b, half2 *c, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	c[i] = __hadd2(a[i], b[i]);
}
__global__ void float22Half2Vec(float2 * src, half2 *des, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	des[i] = __float22half2_rn(src[i]);
 
}
 
__global__ void half22Float2Vec(half2 *src, float2 *des, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	des[i] = __half22float2(src[i]);
	
}
 
int main()
{
	const int blocks = 128;
	const int threads = 128;
	size_t size = blocks*threads * 2;
	float *vec1 = new float[size];
	float *vec2 = new float[size];
	float *res = new float[size];
	for (size_t i = 0; i < size; i++)
	{
		vec2[i] = vec1[i] = i;
	}
	float * vecDev1, *vecDev2, *resDev;
	CHECK(hipMalloc((void **)&vecDev1, size * sizeof(float)));
	CHECK(hipMalloc((void **)&vecDev2, size * sizeof(float)));
	CHECK(hipMalloc((void **)&resDev, size * sizeof(float)));
	CHECK(hipMemcpy(vecDev1, vec1, size * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(vecDev2, vec2, size * sizeof(float), hipMemcpyHostToDevice));

	half2 *vecHalf2Dev1, *vecHalf2Dev2, *resHalf2Dev;
	CHECK(hipMalloc((void **)&vecHalf2Dev1, size * sizeof(float) / 2));
	CHECK(hipMalloc((void **)&vecHalf2Dev2, size * sizeof(float) / 2));
	CHECK(hipMalloc((void **)&resHalf2Dev, size * sizeof(float) / 2));


	float22Half2Vec <<<128, 128 >>> ((float2*)vecDev1, vecHalf2Dev1, size);
	float22Half2Vec <<<128, 128 >>> ((float2*)vecDev2, vecHalf2Dev2, size);
	myHalf2Add <<<128, 128 >>> (vecHalf2Dev1, vecHalf2Dev2, resHalf2Dev, size);
	half22Float2Vec <<<128, 128 >>>(resHalf2Dev, (float2*)resDev, size);

	//half22Float2Vec << <128, 128 >> >(vecHalf2Dev1, (float2*)resDev, size);
	//CHECK(cudaMemcpy(res, resDev, size * sizeof(float), cudaMemcpyDeviceToHost));
	CHECK(hipMemcpy(res, resDev, size * sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < 128; i++)//打印出前64个结果，并与CPU结果对比
	{
		cout << vec1[i] << " + " << vec2[i] << " = " << vec1[i] + vec2[i] << "  ?  " << res[i] << endl;
	}
	for (int i = 128 * 128; i < 128 * 128 + 128; i++)//打印出前64个结果，并与CPU结果对比
	{
		cout << vec1[i] << " + " << vec2[i] << " = " << vec1[i] + vec2[i] << "  ?  " << res[i] << endl;
	}
	delete[] vec1;
	delete[] vec2;
	delete[] res;
	CHECK(hipFree(vecDev1));
	CHECK(hipFree(vecDev2));
	CHECK(hipFree(resDev));
	CHECK(hipFree(vecHalf2Dev1));
	CHECK(hipFree(vecHalf2Dev2));
	CHECK(hipFree(resHalf2Dev));
	system("pause");
	return 0;
}
